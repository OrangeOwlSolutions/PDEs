#include "hip/hip_runtime.h"
#include <stdio.h>
#include <fstream>
#include <iomanip>

// --- Greek pi
#define _USE_MATH_DEFINES
#include <math.h>

#include <hipfft/hipfft.h>

#define BLOCKSIZEX		16
#define BLOCKSIZEY		16

#define prec_save 10

/*******************/
/* iDivUp FUNCTION */
/*******************/
int iDivUp(int a, int b){ return ((a % b) != 0) ? (a / b + 1) : (a / b); }

/********************/
/* CUDA ERROR CHECK */
/********************/
// --- Credit to http://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) { exit(code); }
	}
}

void gpuErrchk(hipError_t ans) { gpuAssert((ans), __FILE__, __LINE__); }

/**************************************************/
/* COMPUTE RIGHT HAND SIDE OF 2D POISSON EQUATION */
/**************************************************/
__global__ void computeRHS(const float * __restrict__ d_x, const float * __restrict__ d_y,
	                       float2 * __restrict__ d_r, const float Lx, const float Ly, const float sigma, 
						   const int M, const int N) {

	const int tidx = threadIdx.x + blockIdx.x * blockDim.x;
	const int tidy = threadIdx.y + blockIdx.y * blockDim.y;

	if ((tidx >= M) || (tidy >= N)) return;

	const float sigmaSquared = sigma * sigma;
	
	const float rSquared = (d_x[tidx] - 0.5f * Lx) * (d_x[tidx] - 0.5f * Lx) +
		                   (d_y[tidy] - 0.5f * Ly) * (d_y[tidy] - 0.5f * Ly);
	
	d_r[tidy * M + tidx].x = expf(-rSquared / (2.f * sigmaSquared)) * (rSquared - 2.f * sigmaSquared) / (sigmaSquared * sigmaSquared);
	d_r[tidy * M + tidx].y = 0.f;

}

/****************************************************/
/* SOLVE 2D POISSON EQUATION IN THE SPECTRAL DOMAIN */
/****************************************************/
__global__ void solvePoisson(const float * __restrict__ d_kx, const float * __restrict__ d_ky, 
	                          float2 * __restrict__ d_r, const int M, const int N)
{
	const int tidx = threadIdx.x + blockIdx.x * blockDim.x;
	const int tidy = threadIdx.y + blockIdx.y * blockDim.y;

	if ((tidx >= M) || (tidy >= N)) return;

	float scale = -(d_kx[tidx] * d_kx[tidx] + d_ky[tidy] * d_ky[tidy]);

	if (tidx == 0 && tidy == 0) scale = 1.f;

	scale = 1.f / scale;
	d_r[M * tidy + tidx].x *= scale;
	d_r[M * tidy + tidx].y *= scale;
	
}

/****************************************************************************/
/* SOLVE 2D POISSON EQUATION IN THE SPECTRAL DOMAIN - SHARED MEMORY VERSION */
/****************************************************************************/
__global__ void solvePoissonShared(const float * __restrict__ d_kx, const float * __restrict__ d_ky,
	float2 * __restrict__ d_r, const int M, const int N)
{
	const int tidx = threadIdx.x + blockIdx.x * blockDim.x;
	const int tidy = threadIdx.y + blockIdx.y * blockDim.y;

	if ((tidx >= M) || (tidy >= N)) return;

	// --- Use shared memory to minimize multiple access to same spectral coordinate values
	__shared__ float kx_s[BLOCKSIZEX], ky_s[BLOCKSIZEY];

	kx_s[threadIdx.x] = d_kx[tidx];
	ky_s[threadIdx.y] = d_ky[tidy];
	__syncthreads();

	float scale = -(kx_s[threadIdx.x] * kx_s[threadIdx.x] + ky_s[threadIdx.y] * ky_s[threadIdx.y]);

	if (tidx == 0 && tidy == 0) scale = 1.f;

	scale = 1.f / scale;
	d_r[M * tidy + tidx].x *= scale;
	d_r[M * tidy + tidx].y *= scale;

}

/******************************/
/* COMPLEX2REAL SCALED KERNEL */
/******************************/
__global__ void complex2RealScaled(float2 * __restrict__ d_r, float * __restrict__ d_result, const int M, const int N, float scale)
{
	const int tidx = threadIdx.x + blockIdx.x * blockDim.x;
	const int tidy = threadIdx.y + blockIdx.y * blockDim.y;

	if ((tidx >= M) || (tidy >= N)) return;

	d_result[tidy * M + tidx] = scale * (d_r[tidy * M + tidx].x - d_r[0].x);
}

/******************************************/
/* COMPLEX2REAL SCALED KERNEL - OPTIMIZED */
/******************************************/
__global__ void complex2RealScaledOptimized(float2 * __restrict__ d_r, float * __restrict__ d_result, const int M, const int N, float scale)
{
	const int tidx = threadIdx.x + blockIdx.x * blockDim.x;
	const int tidy = threadIdx.y + blockIdx.y * blockDim.y;

	if ((tidx >= M) || (tidy >= N)) return;

	__shared__ float d_r_0[1];

	if (threadIdx.x == 0) d_r_0[0] = d_r[0].x;
	
	volatile float2 c;
	c.x = d_r[tidy * M + tidx].x;
	c.y = d_r[tidy * M + tidx].y;

	d_result[tidy * M + tidx] = scale * (c.x - d_r_0[0]);
}

/**************************************/
/* SAVE FLOAT2 ARRAY FROM GPU TO FILE */
/**************************************/
void saveGPUcomplextxt(const float2 * d_in, const char *filename, const int M) {

	float2 *h_in = (float2 *)malloc(M * sizeof(float2));

	gpuErrchk(hipMemcpy(h_in, d_in, M * sizeof(float2), hipMemcpyDeviceToHost));

	std::ofstream outfile;
	outfile.open(filename);
	for (int i = 0; i < M; i++) {
		outfile << std::setprecision(prec_save) << h_in[i].x << "\n"; outfile << std::setprecision(prec_save) << h_in[i].y << "\n";
	}
	outfile.close();

}

/*************************************/
/* SAVE FLOAT ARRAY FROM GPU TO FILE */
/*************************************/
template <class T>
void saveGPUrealtxt(const T * d_in, const char *filename, const int M) {

	T *h_in = (T *)malloc(M * sizeof(T));

	gpuErrchk(hipMemcpy(h_in, d_in, M * sizeof(T), hipMemcpyDeviceToHost));

	std::ofstream outfile;
	outfile.open(filename);
	for (int i = 0; i < M; i++) outfile << std::setprecision(prec_save) << h_in[i] << "\n";
	outfile.close();

}

/********/
/* MAIN */
/********/
int main()
{
	const int	M		= 32;              // --- Number of Fourier harmonics along x (should be a multiple of 2)
	const int	N		= 128;              // --- Number of Fourier harmonics along y (should be a multiple of 2)
	const float Lx		= 3.f;             // --- Domain size along x
	const float Ly		= 1.5f;            // --- Domain size along y
	const float sigma	= 0.1f;            // --- Characteristic width of f (make << 1)
		
	// --- Wavenumbers on the host
	float *h_kx = (float *)malloc(M * sizeof(float));
	float *h_ky = (float *)malloc(N * sizeof(float));
	for (int k = 0; k < M / 2; k++)  h_kx[k]		= (2.f * M_PI / Lx) * k;
	for (int k = -M / 2; k < 0; k++) h_kx[k + M]	= (2.f * M_PI / Lx) * k;
	for (int k = 0; k < N / 2; k++)  h_ky[k]		= (2.f * M_PI / Ly) * k;
	for (int k = -N / 2; k < 0; k++) h_ky[k + N]	= (2.f * M_PI / Ly) * k;

	// --- Wavenumbers on the device
	float *d_kx;	gpuErrchk(hipMalloc(&d_kx, M * sizeof(float)));
	float *d_ky;	gpuErrchk(hipMalloc(&d_ky, N * sizeof(float)));
	gpuErrchk(hipMemcpy(d_kx, h_kx, M * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_ky, h_ky, N * sizeof(float), hipMemcpyHostToDevice));

	// --- Domain discretization on the host
	float *h_x = (float *)malloc(M * sizeof(float));
	float *h_y = (float *)malloc(N * sizeof(float));
	for (int k = 0; k < M; k++)  h_x[k] = Lx / (float)M * k;
	for (int k = 0; k < N; k++)  h_y[k] = Ly / (float)N * k;

	// --- Domain discretization on the device
	float *d_x;		gpuErrchk(hipMalloc(&d_x, M * sizeof(float)));
	float *d_y;		gpuErrchk(hipMalloc(&d_y, N * sizeof(float)));
	gpuErrchk(hipMemcpy(d_x, h_x, M * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_y, h_y, N * sizeof(float), hipMemcpyHostToDevice));

	// --- Compute right-hand side of the differential equation on the device
	float2 *d_r;	gpuErrchk(hipMalloc(&d_r, M * N * sizeof(float2)));
	dim3 dimBlock(BLOCKSIZEX, BLOCKSIZEY);
	dim3 dimGrid(iDivUp(M, BLOCKSIZEX), iDivUp(N, BLOCKSIZEY));
	computeRHS << <dimGrid, dimBlock >> >(d_x, d_y, d_r, Lx, Ly, sigma, M, N);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());

	// --- Create plan for CUDA FFT
	hipfftHandle plan;
	hipfftPlan2d(&plan, N, M, HIPFFT_C2C);
	
	// --- Compute in place forward FFT of right-hand side
	hipfftExecC2C(plan, d_r, d_r, HIPFFT_FORWARD);
	
	// --- Solve Poisson equation in Fourier space 
	//solvePoisson << <dimGrid, dimBlock >> > (d_kx, d_ky, d_r, M, N);
	solvePoissonShared << <dimGrid, dimBlock >> > (d_kx, d_ky, d_r, M, N);

	// --- Compute in place inverse FFT
	hipfftExecC2C(plan, d_r, d_r, HIPFFT_BACKWARD);

	//saveGPUcomplextxt(d_r, ".\d_r.txt", M * N);

	// --- With cuFFT, an FFT followed by an IFFT will return the same array times the size of the transform
	// --- Accordingly, we need to scale the result.
	const float scale = 1.f / ((float)M * (float)N);
	float *d_result;	gpuErrchk(hipMalloc(&d_result, M * N * sizeof(float)));
	//complex2RealScaled << <dimGrid, dimBlock >> > (d_r, d_result, M, N, scale);
	complex2RealScaledOptimized << <dimGrid, dimBlock >> > (d_r, d_result, M, N, scale);

	saveGPUrealtxt(d_result, ".\\d_result.txt", M * N);

	return 0;

}

