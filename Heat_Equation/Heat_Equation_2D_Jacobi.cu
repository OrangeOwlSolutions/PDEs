#include "hip/hip_runtime.h"
#include <iostream>

#include "hip/hip_runtime.h"
#include ""

#include "Utilities.cuh"
#include "InputOutput.cuh"
#include "TimingGPU.cuh"

#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16

#define DEBUG

texture<float, 2, hipReadModeElementType>  tex_T;
texture<float, 2, hipReadModeElementType>  tex_T_old;

/***********************************/
/* JACOBI ITERATION FUNCTION - GPU */
/***********************************/
__global__ void Jacobi_Iterator_GPU(const float * __restrict__ T_old, float * __restrict__ T_new, const int NX, const int NY)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x ;
    const int j = blockIdx.y * blockDim.y + threadIdx.y ;

                                //                         N 
    int P = i + j*NX;           // node (i,j)              |
    int N = i + (j+1)*NX;       // node (i,j+1)            |
    int S = i + (j-1)*NX;       // node (i,j-1)     W ---- P ---- E
    int E = (i+1) + j*NX;       // node (i+1,j)            |
    int W = (i-1) + j*NX;       // node (i-1,j)            |
                                //                         S 

    // --- Only update "interior" (not boundary) node points
	if (i>0 && i<NX-1 && j>0 && j<NY-1) T_new[P] = 0.25 * (T_old[E] + T_old[W] + T_old[N] + T_old[S]); 
}

/******************************************************/
/* JACOBI ITERATION FUNCTION - GPU - SHARED MEMORY V1 */
/******************************************************/
__global__ void Jacobi_Iterator_GPU_shared_v1(const float * __restrict__ T_old, float * __restrict__ T_new, const int NX, const int NY)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x ;
    const int j = blockIdx.y * blockDim.y + threadIdx.y ;

                                //                         N 
    int P = i + j*NX;           // node (i,j)              |
    int N = i + (j+1)*NX;       // node (i,j+1)            |
    int S = i + (j-1)*NX;       // node (i,j-1)     W ---- P ---- E
    int E = (i+1) + j*NX;       // node (i+1,j)            |
    int W = (i-1) + j*NX;       // node (i-1,j)            |
                                //                         S 
	__shared__ float T_sh[BLOCK_SIZE_X][BLOCK_SIZE_Y];

	// --- Load data to shared memory. Halo regions are NOT loaded.
	T_sh[threadIdx.x][threadIdx.y] = T_old[P];
	__syncthreads();

	if ((threadIdx.x > 0) && (threadIdx.x < (BLOCK_SIZE_X - 1)) && (threadIdx.y > 0) && (threadIdx.y < (BLOCK_SIZE_Y ‐ 1))) 
		// --- If we do not need halo region elements, then use shared memory.
		T_new[P] = 0.25 * (T_sh[threadIdx.x][threadIdx.y - 1] + T_sh[threadIdx.x][threadIdx.y + 1] + T_sh[threadIdx.x - 1][threadIdx.y] + T_sh[threadIdx.x + 1][threadIdx.y]);
	else if (i>0 && i<NX-1 && j>0 && j<NY-1)  // --- Only update "interior" (not boundary) node points
		// --- If we need halo region elements, then use global memory.
		T_new[P] = 0.25 * (T_old[E] + T_old[W] + T_old[N] + T_old[S]); 
	
}

/******************************************************/
/* JACOBI ITERATION FUNCTION - GPU - SHARED MEMORY V2 */
/******************************************************/
__global__ void Jacobi_Iterator_GPU_shared_v2(const float * __restrict__ T_old, float * __restrict__ T_new, const int NX, const int NY)
{
    const int i = blockIdx.x * (BLOCK_SIZE_X - 2) + threadIdx.x ;
    const int j = blockIdx.y * (BLOCK_SIZE_Y - 2) + threadIdx.y ;

    int P = i + j*NX;           

	if ((i >= NX) || (j >= NY)) return;
	
	__shared__ float T_sh[BLOCK_SIZE_X][BLOCK_SIZE_Y];

	// --- Load data to shared memory. Halo regions ARE loaded.
	T_sh[threadIdx.x][threadIdx.y] = T_old[P];
	__syncthreads();

	if (((threadIdx.x > 0) && (threadIdx.x < (BLOCK_SIZE_X - 1)) && (threadIdx.y > 0) && (threadIdx.y < (BLOCK_SIZE_Y ‐ 1))) &&
	   (i>0 && i<NX-1 && j>0 && j<NY-1))
		T_new[P] = 0.25 * (T_sh[threadIdx.x][threadIdx.y - 1] + T_sh[threadIdx.x][threadIdx.y + 1] + T_sh[threadIdx.x - 1][threadIdx.y] + T_sh[threadIdx.x + 1][threadIdx.y]);
	
}

/******************************************************/
/* JACOBI ITERATION FUNCTION - GPU - SHARED MEMORY V2 */
/******************************************************/
__global__ void Jacobi_Iterator_GPU_shared_v3(const float * __restrict__ T_old, float * __restrict__ T_new, const int NX, const int NY)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x ;
    const int j = blockIdx.y * blockDim.y + threadIdx.y ;

	const int tid_block = threadIdx.y * BLOCK_SIZE_X + threadIdx.x;		// --- Flattened thread index within a block

	const int i1      = tid_block % (BLOCK_SIZE_X + 2);
	const int j1      = tid_block / (BLOCK_SIZE_Y + 2);

	const int i2      = (BLOCK_SIZE_X * BLOCK_SIZE_Y + tid_block) % (BLOCK_SIZE_X + 2);
	const int j2      = (BLOCK_SIZE_X * BLOCK_SIZE_Y + tid_block) / (BLOCK_SIZE_Y + 2);

    int P = i + j * NX;           

	if ((i >= NX) || (j >= NY)) return;
	
	__shared__ float T_sh[BLOCK_SIZE_X + 2][BLOCK_SIZE_Y + 2];

	if (((blockIdx.x * BLOCK_SIZE_X - 1 + i1) < NX) && ((blockIdx.y * BLOCK_SIZE_Y - 1 + j1) < NY))
		T_sh[i1][j1] = T_old[(blockIdx.x * BLOCK_SIZE_X - 1 + i1) + (blockIdx.y * BLOCK_SIZE_Y - 1 + j1) * NX];

	if (((i2 < (BLOCK_SIZE_X + 2)) && (j2 < (BLOCK_SIZE_Y + 2))) && (((blockIdx.x * BLOCK_SIZE_X - 1 + i2) < NX) && ((blockIdx.y * BLOCK_SIZE_Y - 1 + j2) < NY)))
		T_sh[i2][j2] = T_old[(blockIdx.x * BLOCK_SIZE_X - 1 + i2) + (blockIdx.y * BLOCK_SIZE_Y - 1 + j2) * NX];

	__syncthreads();

	if ((threadIdx.x <= (BLOCK_SIZE_X - 1) && (threadIdx.y <= (BLOCK_SIZE_Y ‐ 1))) && (i>0 && i<NX-1 && j>0 && j<NY-1))
		T_new[P] = 0.25 * (T_sh[threadIdx.x + 1][threadIdx.y] + T_sh[threadIdx.x + 1][threadIdx.y + 2] + T_sh[threadIdx.x][threadIdx.y + 1] + T_sh[threadIdx.x + 2][threadIdx.y + 1]);
	
}

/*********************************************/
/* JACOBI ITERATION FUNCTION - GPU - TEXTURE */
/*********************************************/
__global__ void Jacobi_Iterator_GPU_texture(float * __restrict__ T_new, const bool flag, const int NX, const int NY) {
    
    const int i = blockIdx.x * blockDim.x + threadIdx.x ;
    const int j = blockIdx.y * blockDim.y + threadIdx.y ;

	float P, N, S, E, W;	
	if (flag) {
											//                         N 
        P = tex2D(tex_T_old, i,     j);		// node (i,j)              |
        N = tex2D(tex_T_old, i,     j + 1);	// node (i,j+1)            |
		S = tex2D(tex_T_old, i,     j - 1);	// node (i,j-1)     W ---- P ---- E
        E = tex2D(tex_T_old, i + 1, j);		// node (i+1,j)            |
        W = tex2D(tex_T_old, i - 1, j);		// node (i-1,j)            |
											//                         S 
	} else {
											//                         N 
        P = tex2D(tex_T,     i,     j);		// node (i,j)              |
        N = tex2D(tex_T,     i,     j + 1);	// node (i,j+1)            |
		S = tex2D(tex_T,     i,     j - 1);	// node (i,j-1)     W ---- P ---- E
        E = tex2D(tex_T,     i + 1, j);		// node (i+1,j)            |
        W = tex2D(tex_T,     i - 1, j);		// node (i-1,j)            |
											//                         S 
	}

	// --- Only update "interior" (not boundary) node points
	if (i>0 && i<NX-1 && j>0 && j<NY-1) T_new[i + j*NX] = 0.25 * (E + W + N + S);
}

/***********************************/
/* JACOBI ITERATION FUNCTION - CPU */
/***********************************/
void Jacobi_Iterator_CPU(float * __restrict T, float * __restrict T_new, const int NX, const int NY, const int MAX_ITER)
{
	for(int iter=0; iter<MAX_ITER; iter=iter+2)
    {
	    // --- Only update "interior" (not boundary) node points
        for(int j=1; j<NY-1; j++) 
			for(int i=1; i<NX-1; i++) {
                float T_E = T[(i+1) + NX*j];
                float T_W = T[(i-1) + NX*j];
                float T_N = T[i + NX*(j+1)];
                float T_S = T[i + NX*(j-1)];
                T_new[i+NX*j] = 0.25*(T_E + T_W + T_N + T_S);
            }
 
        for(int j=1; j<NY-1; j++) 
			for(int i=1; i<NX-1; i++) {
                float T_E = T_new[(i+1) + NX*j];
                float T_W = T_new[(i-1) + NX*j];
                float T_N = T_new[i + NX*(j+1)];
                float T_S = T_new[i + NX*(j-1)];
                T[i+NX*j] = 0.25*(T_E + T_W + T_N + T_S);
            }
    }
}

/******************************/
/* TEMPERATURE INITIALIZATION */
/******************************/
void Initialize(float * __restrict h_T, const int NX, const int NY)
{
    // --- Set left wall to 1
    for(int j=0; j<NY; j++) h_T[j * NX] = 1.0;
}


/********/
/* MAIN */
/********/
int main()
{
	const int NX = 256;			// --- Number of discretization points along the x axis
	const int NY = 256;			// --- Number of discretization points along the y axis

	const int MAX_ITER = 100;	// --- Number of Jacobi iterations

    // --- CPU temperature distributions
    float *h_T				= (float *)calloc(NX * NY, sizeof(float));
    float *h_T_old			= (float *)calloc(NX * NY, sizeof(float));
    Initialize(h_T,     NX, NY);
    Initialize(h_T_old, NX, NY);
    float *h_T_GPU_result		= (float *)malloc(NX * NY * sizeof(float));
    float *h_T_GPU_tex_result	= (float *)malloc(NX * NY * sizeof(float));
    float *h_T_GPU_sh1_result	= (float *)malloc(NX * NY * sizeof(float));
    float *h_T_GPU_sh2_result	= (float *)malloc(NX * NY * sizeof(float));
    float *h_T_GPU_sh3_result	= (float *)malloc(NX * NY * sizeof(float));

    // --- GPU temperature distribution
    float *d_T;			gpuErrchk(hipMalloc((void**)&d_T,			NX * NY * sizeof(float)));
    float *d_T_old;		gpuErrchk(hipMalloc((void**)&d_T_old,		NX * NY * sizeof(float)));
    float *d_T_tex;		gpuErrchk(hipMalloc((void**)&d_T_tex,		NX * NY * sizeof(float)));
    float *d_T_old_tex;	gpuErrchk(hipMalloc((void**)&d_T_old_tex,	NX * NY * sizeof(float)));
    float *d_T_sh1;		gpuErrchk(hipMalloc((void**)&d_T_sh1,		NX * NY * sizeof(float)));
    float *d_T_old_sh1;	gpuErrchk(hipMalloc((void**)&d_T_old_sh1,	NX * NY * sizeof(float)));
    float *d_T_sh2;		gpuErrchk(hipMalloc((void**)&d_T_sh2,		NX * NY * sizeof(float)));
    float *d_T_old_sh2;	gpuErrchk(hipMalloc((void**)&d_T_old_sh2,	NX * NY * sizeof(float)));
    float *d_T_sh3;		gpuErrchk(hipMalloc((void**)&d_T_sh3,		NX * NY * sizeof(float)));
    float *d_T_old_sh3;	gpuErrchk(hipMalloc((void**)&d_T_old_sh3,	NX * NY * sizeof(float)));

    gpuErrchk(hipMemcpy(d_T,			h_T,	 NX * NY * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_T_tex,		h_T,	 NX * NY * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_T_sh1,		h_T,	 NX * NY * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_T_sh2,		h_T,	 NX * NY * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_T_sh3,		h_T,	 NX * NY * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_T_old,		d_T,	 NX * NY * sizeof(float), hipMemcpyDeviceToDevice));
    gpuErrchk(hipMemcpy(d_T_old_tex,	d_T_tex, NX * NY * sizeof(float), hipMemcpyDeviceToDevice));
    gpuErrchk(hipMemcpy(d_T_old_sh1,	d_T_sh1, NX * NY * sizeof(float), hipMemcpyDeviceToDevice));
    gpuErrchk(hipMemcpy(d_T_old_sh2,	d_T_sh2, NX * NY * sizeof(float), hipMemcpyDeviceToDevice));
    gpuErrchk(hipMemcpy(d_T_old_sh3,	d_T_sh3, NX * NY * sizeof(float), hipMemcpyDeviceToDevice));

	//hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
	hipChannelFormatDesc desc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

    gpuErrchk(hipBindTexture2D(NULL, &tex_T,	  d_T_tex,     &desc, NX, NY, sizeof(float) * NX));
    gpuErrchk(hipBindTexture2D(NULL, &tex_T_old, d_T_old_tex, &desc, NX, NY, sizeof(float) * NX));

	tex_T.addressMode[0] = hipAddressModeWrap;
	tex_T.addressMode[1] = hipAddressModeWrap;
	tex_T.filterMode = hipFilterModePoint;
	tex_T.normalized = false;
	
	tex_T_old.addressMode[0] = hipAddressModeWrap;
	tex_T_old.addressMode[1] = hipAddressModeWrap;
	tex_T_old.filterMode = hipFilterModePoint;
	tex_T_old.normalized = false;

	// --- Grid size
    dim3 dimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y);
    dim3 dimGrid (iDivUp(NX, BLOCK_SIZE_X), iDivUp(NY, BLOCK_SIZE_Y));

    // --- Jacobi iterations on the host
	Jacobi_Iterator_CPU(h_T, h_T_old, NX, NY, MAX_ITER);

	// --- Jacobi iterations on the device
	TimingGPU timerGPU;
	timerGPU.StartCounter();
	for (int k=0; k<MAX_ITER; k=k+2) {
        Jacobi_Iterator_GPU<<<dimGrid, dimBlock>>>(d_T,     d_T_old, NX, NY);   // --- Update d_T_old     starting from data stored in d_T
#ifdef DEBUG
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif        
		Jacobi_Iterator_GPU<<<dimGrid, dimBlock>>>(d_T_old, d_T    , NX, NY);   // --- Update d_T         starting from data stored in d_T_old
 #ifdef DEBUG
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif
	}
	printf("Timing = %f ms\n", timerGPU.GetCounter());

	// --- Jacobi iterations on the device - shared memory v1
	timerGPU.StartCounter();
	for (int k=0; k<MAX_ITER; k=k+2) {
        Jacobi_Iterator_GPU_shared_v1<<<dimGrid, dimBlock>>>(d_T_sh1,     d_T_old_sh1, NX, NY);   // --- Update d_T_old     starting from data stored in d_T
#ifdef DEBUG
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif        
        Jacobi_Iterator_GPU_shared_v1<<<dimGrid, dimBlock>>>(d_T_old_sh1, d_T_sh1    , NX, NY);   // --- Update d_T         starting from data stored in d_T_old
#ifdef DEBUG
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif        
    }
	printf("Timing with shared memory v1 = %f ms\n", timerGPU.GetCounter());

	// --- Jacobi iterations on the device - shared memory v2
    dim3 dimBlock2(BLOCK_SIZE_X, BLOCK_SIZE_Y);
    dim3 dimGrid2 (iDivUp(NX, BLOCK_SIZE_X - 2), iDivUp(NY, BLOCK_SIZE_Y - 2));
	timerGPU.StartCounter();
	for (int k=0; k<MAX_ITER; k=k+2) {
        Jacobi_Iterator_GPU_shared_v2<<<dimGrid2, dimBlock>>>(d_T_sh2,     d_T_old_sh2, NX, NY);   // --- Update d_T_old     starting from data stored in d_T
#ifdef DEBUG
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif        
        Jacobi_Iterator_GPU_shared_v2<<<dimGrid2, dimBlock>>>(d_T_old_sh2, d_T_sh2    , NX, NY);   // --- Update d_T         starting from data stored in d_T_old
#ifdef DEBUG
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif        
    }
	printf("Timing with shared memory v2 = %f ms\n", timerGPU.GetCounter());

	// --- Jacobi iterations on the device - shared memory v3
	timerGPU.StartCounter();
	for (int k=0; k<MAX_ITER; k=k+2) {
        Jacobi_Iterator_GPU_shared_v3<<<dimGrid, dimBlock>>>(d_T_sh3,     d_T_old_sh3, NX, NY);   // --- Update d_T_old     starting from data stored in d_T
#ifdef DEBUG
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif        
        Jacobi_Iterator_GPU_shared_v3<<<dimGrid, dimBlock>>>(d_T_old_sh3, d_T_sh3    , NX, NY);   // --- Update d_T         starting from data stored in d_T_old
#ifdef DEBUG
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif        
    }
	printf("Timing with shared memory v3 = %f ms\n", timerGPU.GetCounter());

	// --- Jacobi iterations on the device - texture case
	timerGPU.StartCounter();
	for (int k=0; k<MAX_ITER; k=k+2) {
        Jacobi_Iterator_GPU_texture<<<dimGrid, dimBlock>>>(d_T_old_tex, 0, NX, NY);   // --- Update d_T_tex         starting from data stored in d_T_old_tex
#ifdef DEBUG
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif        
        Jacobi_Iterator_GPU_texture<<<dimGrid, dimBlock>>>(d_T_tex,     1, NX, NY);   // --- Update d_T_old_tex     starting from data stored in d_T_tex
#ifdef DEBUG
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif        
    }
	printf("Timing with texture = %f ms\n", timerGPU.GetCounter());

	saveCPUrealtxt(h_T,     "C:\\Users\\Documents\\Project\\Differential_Equations\\Heat_Equation\\2D\\DiffusionEquationJacobi\\DiffusionEquation\\CPU_result.txt",     NX * NY);
	saveGPUrealtxt(d_T_tex, "C:\\Users\\Documents\\Project\\Differential_Equations\\Heat_Equation\\2D\\DiffusionEquationJacobi\\DiffusionEquation\\GPU_result_tex.txt", NX * NY);
	saveGPUrealtxt(d_T,     "C:\\Users\\Documents\\Project\\Differential_Equations\\Heat_Equation\\2D\\DiffusionEquationJacobi\\DiffusionEquation\\GPU_result.txt",     NX * NY);
	saveGPUrealtxt(d_T_sh1, "C:\\Users\\Documents\\Project\\Differential_Equations\\Heat_Equation\\2D\\DiffusionEquationJacobi\\DiffusionEquation\\GPU_result_sh1.txt",     NX * NY);
	saveGPUrealtxt(d_T_sh2, "C:\\Users\\Documents\\Project\\Differential_Equations\\Heat_Equation\\2D\\DiffusionEquationJacobi\\DiffusionEquation\\GPU_result_sh2.txt",     NX * NY);
	saveGPUrealtxt(d_T_sh3, "C:\\Users\\Documents\\Project\\Differential_Equations\\Heat_Equation\\2D\\DiffusionEquationJacobi\\DiffusionEquation\\GPU_result_sh3.txt",     NX * NY);
	
	// --- Copy results from device to host
    gpuErrchk(hipMemcpy(h_T_GPU_result,	 d_T,	  NX * NY * sizeof(float), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_T_GPU_tex_result, d_T_tex, NX * NY * sizeof(float), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_T_GPU_sh1_result, d_T_sh1, NX * NY * sizeof(float), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_T_GPU_sh2_result, d_T_sh2, NX * NY * sizeof(float), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_T_GPU_sh3_result, d_T_sh3, NX * NY * sizeof(float), hipMemcpyDeviceToHost));
	
	// --- Calculate percentage root mean square error between host and device results
	float sum = 0.f, sum_tex = 0.f, sum_ref = 0.f, sum_sh1 = 0.f, sum_sh2 = 0.f, sum_sh3 = 0.f;
	for (int j=0; j<NY; j++)
		for (int i=0; i<NX; i++) {
			sum     = sum     + (h_T_GPU_result    [j * NX + i] - h_T[j * NX + i]) * (h_T_GPU_result    [j * NX + i] - h_T[j * NX + i]);
			sum_tex = sum_tex + (h_T_GPU_tex_result[j * NX + i] - h_T[j * NX + i]) * (h_T_GPU_tex_result[j * NX + i] - h_T[j * NX + i]);
			sum_sh1 = sum_sh1 + (h_T_GPU_sh1_result[j * NX + i] - h_T[j * NX + i]) * (h_T_GPU_sh1_result[j * NX + i] - h_T[j * NX + i]);
			sum_sh2 = sum_sh2 + (h_T_GPU_sh2_result[j * NX + i] - h_T[j * NX + i]) * (h_T_GPU_sh2_result[j * NX + i] - h_T[j * NX + i]);
			sum_sh3 = sum_sh3 + (h_T_GPU_sh3_result[j * NX + i] - h_T[j * NX + i]) * (h_T_GPU_sh3_result[j * NX + i] - h_T[j * NX + i]);
			sum_ref = sum_ref + h_T[j * NX + i]								   * h_T[j * NX + i];
		}
	printf("Percentage root mean square error           = %f\n", 100.*sqrt(sum     / sum_ref));
	printf("Percentage root mean square error texture   = %f\n", 100.*sqrt(sum_tex / sum_ref));
	printf("Percentage root mean square error shared v1 = %f\n", 100.*sqrt(sum_sh1 / sum_ref));
	printf("Percentage root mean square error shared v2 = %f\n", 100.*sqrt(sum_sh2 / sum_ref));
	printf("Percentage root mean square error shared v3 = %f\n", 100.*sqrt(sum_sh3 / sum_ref));
	
    return 0;
}
