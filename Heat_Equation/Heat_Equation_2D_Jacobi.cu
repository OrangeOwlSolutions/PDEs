#include "hip/hip_runtime.h"
#include <iostream>

#include "hip/hip_runtime.h"
#include ""

#include "Utilities.cuh"
#include "InputOutput.cuh"
#include "TimingGPU.cuh"

#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16

texture<float, 2, hipReadModeElementType>  tex_T;
texture<float, 2, hipReadModeElementType>  tex_T_old;

/***********************************/
/* JACOBI ITERATION FUNCTION - GPU */
/***********************************/
__global__ void Jacobi_Iterator_GPU(const float * __restrict__ T_old, float * __restrict__ T_new, const int NX, const int NY)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x ;
    const int j = blockIdx.y * blockDim.y + threadIdx.y ;

                                //                         N 
    int P = i + j*NX;           // node (i,j)              |
    int N = i + (j+1)*NX;       // node (i,j+1)            |
    int S = i + (j-1)*NX;       // node (i,j-1)     W ---- P ---- E
    int E = (i+1) + j*NX;       // node (i+1,j)            |
    int W = (i-1) + j*NX;       // node (i-1,j)            |
                                //                         S 

    // --- Only update "interior" (not boundary) node points
	if (i>0 && i<NX-1 && j>0 && j<NY-1) T_new[P] = 0.25 * (T_old[E] + T_old[W] + T_old[N] + T_old[S]); 
}

/*********************************************/
/* JACOBI ITERATION FUNCTION - GPU - TEXTURE */
/*********************************************/
__global__ void Jacobi_Iterator_GPU_texture(float * __restrict__ T_new, const bool flag, const int NX, const int NY) {
    
    const int i = blockIdx.x * blockDim.x + threadIdx.x ;
    const int j = blockIdx.y * blockDim.y + threadIdx.y ;

	float P, N, S, E, W;	
	if (flag) {
											//                         N 
        P = tex2D(tex_T_old, i,     j);		// node (i,j)              |
        N = tex2D(tex_T_old, i,     j + 1);	// node (i,j+1)            |
		S = tex2D(tex_T_old, i,     j - 1);	// node (i,j-1)     W ---- P ---- E
        E = tex2D(tex_T_old, i + 1, j);		// node (i+1,j)            |
        W = tex2D(tex_T_old, i - 1, j);		// node (i-1,j)            |
											//                         S 
	} else {
											//                         N 
        P = tex2D(tex_T,     i,     j);		// node (i,j)              |
        N = tex2D(tex_T,     i,     j + 1);	// node (i,j+1)            |
		S = tex2D(tex_T,     i,     j - 1);	// node (i,j-1)     W ---- P ---- E
        E = tex2D(tex_T,     i + 1, j);		// node (i+1,j)            |
        W = tex2D(tex_T,     i - 1, j);		// node (i-1,j)            |
											//                         S 
	}

	// --- Only update "interior" (not boundary) node points
	if (i>0 && i<NX-1 && j>0 && j<NY-1) T_new[i + j*NX] = 0.25 * (E + W + N + S);
}

/***********************************/
/* JACOBI ITERATION FUNCTION - CPU */
/***********************************/
void Jacobi_Iterator_CPU(float * __restrict T, float * __restrict T_new, const int NX, const int NY, const int MAX_ITER)
{
	for(int iter=0; iter<MAX_ITER; iter=iter+2)
    {
	    // --- Only update "interior" (not boundary) node points
        for(int j=1; j<NY-1; j++) 
			for(int i=1; i<NX-1; i++) {
                float T_E = T[(i+1) + NX*j];
                float T_W = T[(i-1) + NX*j];
                float T_N = T[i + NX*(j+1)];
                float T_S = T[i + NX*(j-1)];
                T_new[i+NX*j] = 0.25*(T_E + T_W + T_N + T_S);
            }
 
        for(int j=1; j<NY-1; j++) 
			for(int i=1; i<NX-1; i++) {
                float T_E = T_new[(i+1) + NX*j];
                float T_W = T_new[(i-1) + NX*j];
                float T_N = T_new[i + NX*(j+1)];
                float T_S = T_new[i + NX*(j-1)];
                T[i+NX*j] = 0.25*(T_E + T_W + T_N + T_S);
            }
    }
}

/******************************/
/* TEMPERATURE INITIALIZATION */
/******************************/
void Initialize(float * __restrict h_T, const int NX, const int NY)
{
    // --- Set left wall to 1
    for(int j=0; j<NY; j++) h_T[j * NX] = 1.0;
}


/********/
/* MAIN */
/********/
int main()
{
	const int NX = 1024;			// --- Number of discretization points along the x axis
	const int NY = 1024;			// --- Number of discretization points along the y axis

	const int MAX_ITER = 1;	// --- Number of Jacobi iterations

    // --- CPU temperature distributions
    float *h_T				= (float *)calloc(NX * NY, sizeof(float));
    float *h_T_old			= (float *)calloc(NX * NY, sizeof(float));
    Initialize(h_T,     NX, NY);
    Initialize(h_T_old, NX, NY);
    float *h_T_GPU_result	= (float *)malloc(NX * NY * sizeof(float));

    // --- GPU temperature distribution
    float *d_T;			gpuErrchk(hipMalloc((void**)&d_T,			NX * NY * sizeof(float)));
    float *d_T_old;		gpuErrchk(hipMalloc((void**)&d_T_old,		NX * NY * sizeof(float)));
    float *d_T_tex;		gpuErrchk(hipMalloc((void**)&d_T_tex,		NX * NY * sizeof(float)));
    float *d_T_old_tex;	gpuErrchk(hipMalloc((void**)&d_T_old_tex,	NX * NY * sizeof(float)));

    gpuErrchk(hipMemcpy(d_T,			h_T,	 NX * NY * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_T_tex,		h_T,	 NX * NY * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_T_old,		d_T,	 NX * NY * sizeof(float), hipMemcpyDeviceToDevice));
    gpuErrchk(hipMemcpy(d_T_old_tex,	d_T_tex, NX * NY * sizeof(float), hipMemcpyDeviceToDevice));

    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
    gpuErrchk(hipBindTexture2D(NULL, tex_T,	 d_T_tex,     desc, NX, NY, sizeof(float) * NX));
    gpuErrchk(hipBindTexture2D(NULL, tex_T_old, d_T_old_tex, desc, NX, NY, sizeof(float) * NX));

	// --- Grid size
    dim3 dimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y);
    dim3 dimGrid (iDivUp(NX, BLOCK_SIZE_X), iDivUp(NY, BLOCK_SIZE_Y));

    // --- Jacobi iterations on the host
	Jacobi_Iterator_CPU(h_T, h_T_old, NX, NY, MAX_ITER);

	// --- Jacobi iterations on the device
	TimingGPU timerGPU;
	timerGPU.StartCounter();
	for (int k=0; k<MAX_ITER; k=k+2) {
        Jacobi_Iterator_GPU<<<dimGrid, dimBlock>>>(d_T,     d_T_old, NX, NY);   // --- Update d_T_old     starting from data stored in d_T
        Jacobi_Iterator_GPU<<<dimGrid, dimBlock>>>(d_T_old, d_T    , NX, NY);   // --- Update d_T         starting from data stored in d_T_old
    }
	printf("Timing with no texture = %f ms\n", timerGPU.GetCounter());

	// --- Jacobi iterations on the device - texture case
	timerGPU.StartCounter();
	for (int k=0; k<MAX_ITER; k=k+2) {
        Jacobi_Iterator_GPU_texture<<<dimGrid, dimBlock>>>(d_T_tex,     1, NX, NY);   // --- Update d_T_old_tex     starting from data stored in d_T_tex
        Jacobi_Iterator_GPU_texture<<<dimGrid, dimBlock>>>(d_T_old_tex, 0, NX, NY);   // --- Update d_T_tex         starting from data stored in d_T_old_tex
    }
	printf("Timing with texture = %f ms\n", timerGPU.GetCounter());

	saveCPUrealtxt(h_T,     "C:\\Users\\angelo\\Documents\\Project\\Corsi\\Differential_Equations\\Heat_Equation\\2D\\DiffusionEquationJacobi\\DiffusionEquation\\CPU_result.txt",     NX * NY);
	saveGPUrealtxt(d_T_tex, "C:\\Users\\angelo\\Documents\\Project\\Corsi\\Differential_Equations\\Heat_Equation\\2D\\DiffusionEquationJacobi\\DiffusionEquation\\GPU_result_tex.txt", NX * NY);
	saveGPUrealtxt(d_T,     "C:\\Users\\angelo\\Documents\\Project\\Corsi\\Differential_Equations\\Heat_Equation\\2D\\DiffusionEquationJacobi\\DiffusionEquation\\GPU_result.txt",     NX * NY);
	
	// --- Copy result from device to host
    gpuErrchk(hipMemcpy(h_T_GPU_result, d_T, NX * NY * sizeof(float), hipMemcpyDeviceToHost));

	// --- Calculate percentage root mean square error between host and device results
	float sum = 0., sum_ref = 0.;
	for (int j=0; j<NY; j++)
		for (int i=0; i<NX; i++) {
			sum     = sum     + (h_T_GPU_result[j * NX + i] - h_T[j * NX + i]) * (h_T_GPU_result[j * NX + i] - h_T[j * NX + i]);
			sum_ref = sum_ref + h_T[j * NX + i]								   * h_T[j * NX + i];
		}
	printf("Percentage root mean square error = %f\n", 100.*sqrt(sum / sum_ref));
	
	// --- Release host memory 
    free(h_T);
	free(h_T_GPU_result);

    // --- Release device memory
    gpuErrchk(hipFree(d_T));
    gpuErrchk(hipFree(d_T_old));

    return 0;
}
