#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <thrust\device_vector.h>

#include "Utilities.cuh"

#define BLOCKSIZE  512

/****************************/
/* CPU CALCULATION FUNCTION */
/****************************/
void HeatEquation1DCPU(float * __restrict__ h_T, int *Niter, const float T0, const float Q_N_1, const float dx, const float k, const float rho, 
					   const float cp, const float alpha, const float dt, const float maxErr, const int maxIterNumber, const int N)
{
    float *h_DeltaT = (float *)malloc(N * sizeof(float));

    // --- Enforcing boundary condition at the left end.
    *h_T = T0;
    h_DeltaT[0] = 0.f;

    float current_max;
	do {
        // --- Internal region between the two boundaries.
        for (int i = 1; i < N - 1; i++) h_DeltaT[i] = dt * alpha * ((h_T[i - 1] + h_T[i + 1] - 2.f * h_T[i]) / (dx * dx));

        // --- Enforcing boundary condition at the right end.
        h_DeltaT[N - 1] = dt * 2.f * ((k * ((h_T[N - 2] - h_T[N - 1]) / dx) + Q_N_1) / (dx * rho * cp));

		// --- Update the temperature and find the maximum DeltaT over all nodes
		current_max = h_DeltaT[0]; // --- Remember: h_DeltaT[0] = 0
        for (int i = 1; i < N; i++)
        {
            h_T[i] = h_T[i] + h_DeltaT[i]; // h_T[0] keeps
            current_max = abs(h_DeltaT[i]) > current_max ? abs(h_DeltaT[i]) : current_max;
        }

        // --- Increase iteration counter
        (*Niter)++;

    } while (*Niter < maxIterNumber && current_max > maxErr);

    delete [] h_DeltaT;
}

/**************************/
/* GPU CALCULATION KERNEL */
/**************************/
__global__ void HeatEquation1DGPU_IterationKernel(float * __restrict__ d_T, float * __restrict__ d_DeltaT, const float T0, const float Q_N_1, const float dx, const float k, const float rho, 
					   const float cp, const float alpha, const float dt, const float maxErr, const int maxIterNumber, const int N)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (tid < N) {
		
        // --- Internal region between the two boundaries.
		if ((tid > 0) && (tid < N - 1) ) d_DeltaT[tid]  = dt * alpha *((d_T[tid - 1] + d_T[tid + 1] - 2.f * d_T[tid]) / (dx * dx));
		// --- Enforcing boundary condition at the left end.
		if (tid == 0)					 d_DeltaT[0]	= 0.f;
		// --- Enforcing boundary condition at the right end.
 		if (tid == N - 1)				 d_DeltaT[tid]	= dt * 2.f * ((k * ((d_T[tid - 1] - d_T[tid]) / dx) + Q_N_1) / (dx * rho * cp));
		
		// --- Update the temperature
        d_T[tid] = d_T[tid] + d_DeltaT[tid]; 

		d_DeltaT[tid] = abs(d_DeltaT[tid]);
	}
	
}

__global__ void HeatEquation1DGPU_IterationSharedKernel(float * __restrict__ d_T, float * __restrict__ d_DeltaT, const float T0, const float Q_N_1, const float dx, const float k, const float rho, 
					   const float cp, const float alpha, const float dt, const float maxErr, const int maxIterNumber, const int N)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
	
	// --- Shared memory has 0, 1, ..., BLOCKSIZE - 1, BLOCKSIZE locations, so it has BLOCKSIZE locations + 2 (left and right) halo cells.
	__shared__ float d_T_shared[BLOCKSIZE + 2];				// --- Need to know BLOCKSIZE beforehand
	
	if (tid < N) {
		
		// --- Load data from global memory to shared memory locations 1, 2, ..., BLOCKSIZE - 1
		d_T_shared[threadIdx.x + 1] = d_T[tid];					

		// --- Left halo cell
		if ((threadIdx.x == 0) && (tid > 0)) { d_T_shared[0] = d_T[tid - 1]; }			

		// --- Right halo cell
		if ((threadIdx.x == blockDim.x - 1) && (tid < N - 1)) { d_T_shared[threadIdx.x + 2] = d_T[tid + 1]; } 

		__syncthreads();
		
		// --- Internal region between the two boundaries.
		if ((tid > 0) && (tid < N - 1) ) d_DeltaT[tid]  = dt * alpha *((d_T_shared[threadIdx.x] + d_T_shared[threadIdx.x + 2] - 2.f * d_T_shared[threadIdx.x + 1]) / (dx * dx));
		
		// --- Enforcing boundary condition at the left end.
		if (tid == 0)					 d_DeltaT[0]	= 0.f;
		
		// --- Enforcing boundary condition at the right end.
 		if (tid == N - 1)				 d_DeltaT[tid]	= dt * 2.f * ((k * ((d_T_shared[threadIdx.x] - d_T_shared[threadIdx.x + 1]) / dx) + Q_N_1) / (dx * rho * cp));
		
		// --- Update the temperature
        d_T[tid] = d_T[tid] + d_DeltaT[tid]; 

		d_DeltaT[tid] = abs(d_DeltaT[tid]);
	}
	
}

/****************************/
/* GPU CALCULATION FUNCTION */
/****************************/
void HeatEquation1DGPU(float * __restrict__ d_T, int *Niter, const float T0, const float Q_N_1, const float dx, const float k, const float rho, 
					   const float cp, const float alpha, const float dt, const float maxErr, const int maxIterNumber, const int N)
{
	// --- Absolute values of DeltaT
	float *d_DeltaT;	gpuErrchk(hipMalloc(&d_DeltaT, N * sizeof(float)));

    // --- Enforcing boundary condition at the left end.
    gpuErrchk(hipMemcpy(d_T, &T0, sizeof(float), hipMemcpyHostToDevice));

	float current_max = 0.f;
	do {
		//HeatEquation1DGPU_IterationKernel<<<iDivUp(N, BLOCKSIZE), BLOCKSIZE>>>(d_T, d_DeltaT, T0, Q_N_1, dx, k, rho, cp, alpha, dt, maxErr, maxIterNumber, N);
		HeatEquation1DGPU_IterationSharedKernel<<<iDivUp(N, BLOCKSIZE), BLOCKSIZE>>>(d_T, d_DeltaT, T0, Q_N_1, dx, k, rho, cp, alpha, dt, maxErr, maxIterNumber, N);

		thrust::device_ptr<float> d = thrust::device_pointer_cast(d_DeltaT);  
		current_max = thrust::reduce(d, d + N, current_max, thrust::maximum<float>());

        // --- Increase iteration counter
		(*Niter)++;

    } while (*Niter < maxIterNumber && current_max > maxErr);

    gpuErrchk(hipFree(d_DeltaT));
}

/********/
/* MAIN */
/********/
int main()
{
	// --- See https://en.wikipedia.org/wiki/Thermal_diffusivity
	
	// --- Parameters of the problem
    const float k			= 0.19f;					// --- Thermal conductivity [W / (m * K)]
    const float rho			= 930.f;					// --- Density [kg / m^3]
    const float cp			= 1340.f;					// --- Specific heat capacity [J / (kg * K)]
    const float alpha		= k / (rho * cp);			// --- Thermal diffusivity [m^2 / s]
	const float length		= 1.6f;						// --- Total length of the domain [m]
	const int N				= 64 * BLOCKSIZE;			// --- Number of grid points
	const float dx			= (length / (float)(N - 1));// --- Discretization step [m]
    const float dt			= (float)(dx * dx / (4.f * alpha));
														// --- Time step [s]
	const float T0			= 0.f;						// --- Temperature at the first end of the domain [C]
    const float Q_N_1		= 10.f;						// --- Heat flux at the second end of the domain [W / m^2]
    const float maxErr		= 1.0e-5f;					// --- Maximum admitted DeltaT
    const int maxIterNumber = 10.0 / dt;				// --- Number of overall time steps

    /********************/
	/* GPU CALCULATIONS */
    /********************/
	float *h_T_final_device = (float *)malloc(N * sizeof(float));		// --- Final "host-side" result of GPU calculations
    int Niter_GPU = 0;													// --- Iteration counter for GPU calculations

    // --- Device temperature allocation and initialization
	float *d_T;		gpuErrchk(hipMalloc(&d_T, N * sizeof(float)));
    gpuErrchk(hipMemset(d_T, 0, N * sizeof(float))); 

    // --- GPU calculations
	HeatEquation1DGPU(d_T, &Niter_GPU, T0, Q_N_1, dx, k, rho, cp, alpha, dt, maxErr, maxIterNumber, N);

    // --- Transfer the GPU calculation results from device to host
    gpuErrchk(hipMemcpy(h_T_final_device, d_T, N * sizeof(float), hipMemcpyDeviceToHost));

    /********************/
	/* CPU CALCULATIONS */
    /********************/
    // --- Host temperature allocation and initialization
    float *h_T_final_host = (float *)malloc(N * sizeof(float));
    memset(h_T_final_host, 0, N * sizeof(float));
    
	int Niter_CPU = 0;

    HeatEquation1DCPU(h_T_final_host, &Niter_CPU, T0, Q_N_1, dx, k, rho, cp, alpha, dt, maxErr, maxIterNumber, N);

    /************************/
	/* CHECKING THE RESULTS */
    /************************/
	for (int i = 0; i < N; i++) {
        printf("Node = %i; T_host = %3.10f; T_device = %3.10f\n", i, h_T_final_host[i], h_T_final_device[i]);
		if (h_T_final_host[i] != h_T_final_device[i]) {
            printf("Error at i = %i; T_host = %f; T_device = %f\n", i, h_T_final_host[i], h_T_final_device[i]);
            return 0;
        }
    }

    printf("Test passed!\n");

    delete [] h_T_final_device;
    gpuErrchk(hipFree(d_T));

    return 0;
}
