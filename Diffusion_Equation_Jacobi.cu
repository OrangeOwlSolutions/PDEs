#include "hip/hip_runtime.h"
#include <iostream>

#include "hip/hip_runtime.h"
#include ""

#include "Utilities.cuh"

#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16

/***********************************/
/* JACOBI ITERATION FUNCTION - GPU */
/***********************************/
__global__ void Jacobi_Iterator_GPU(const float * __restrict__ T_old, float * __restrict__ T_new, const int NX, const int NY)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x ;
    const int j = blockIdx.y * blockDim.y + threadIdx.y ;

                                //                         N 
    int P = i + j*NX;           // node (i,j)              |
    int N = i + (j+1)*NX;       // node (i,j+1)            |
    int S = i + (j-1)*NX;       // node (i,j-1)     W ---- P ---- E
    int E = (i+1) + j*NX;       // node (i+1,j)            |
    int W = (i-1) + j*NX;       // node (i-1,j)            |
                                //                         S 

    // --- Only update "interior" (not boundary) node points
    if (i>0 && i<NX-1 && j>0 && j<NY-1) T_new[P] = 0.25 * (T_old[E] + T_old[W] + T_old[N] + T_old[S]);
}

/***********************************/
/* JACOBI ITERATION FUNCTION - CPU */
/***********************************/
void Jacobi_Iterator_CPU(float * __restrict T, float * __restrict T_new, const int NX, const int NY, const int MAX_ITER)
{
	for(int iter=0; iter<MAX_ITER; iter=iter+2)
    {
	    // --- Only update "interior" (not boundary) node points
        for(int j=1; j<NY-1; j++) 
			for(int i=1; i<NX-1; i++) {
                float T_E = T[(i+1) + NX*j];
                float T_W = T[(i-1) + NX*j];
                float T_N = T[i + NX*(j+1)];
                float T_S = T[i + NX*(j-1)];
                T_new[i+NX*j] = 0.25*(T_E + T_W + T_N + T_S);
            }
 
        for(int j=1; j<NY-1; j++) 
			for(int i=1; i<NX-1; i++) {
                float T_E = T_new[(i+1) + NX*j];
                float T_W = T_new[(i-1) + NX*j];
                float T_N = T_new[i + NX*(j+1)];
                float T_S = T_new[i + NX*(j-1)];
                T[i+NX*j] = 0.25*(T_E + T_W + T_N + T_S);
            }
    }
}

/******************************/
/* TEMPERATURE INITIALIZATION */
/******************************/
void Initialize(float * __restrict h_T, const int NX, const int NY)
{
    // --- Set left wall to 1
    for(int j=0; j<NY; j++) h_T[j * NX] = 1.0;
}


/********/
/* MAIN */
/********/
int main()
{
	const int NX = 256;			// --- Number of discretization points along the x axis
	const int NY = 256;			// --- Number of discretization points along the y axis

	const int MAX_ITER = 1;		// --- Number of Jacobi iterations

    // --- CPU temperature distributions
    float *h_T				= (float *)calloc(NX * NY, sizeof(float));
    float *h_T_old			= (float *)calloc(NX * NY, sizeof(float));
    Initialize(h_T,     NX, NY);
    Initialize(h_T_old, NX, NY);
    float *h_T_GPU_result	= (float *)malloc(NX * NY * sizeof(float));

    // --- GPU temperature distribution
    float *d_T;		gpuErrchk(hipMalloc((void**)&d_T,		NX * NY * sizeof(float)));
    float *d_T_old;	gpuErrchk(hipMalloc((void**)&d_T_old,	NX * NY * sizeof(float)));

    gpuErrchk(hipMemcpy(d_T,     h_T, NX * NY * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_T_old, d_T, NX * NY * sizeof(float), hipMemcpyDeviceToDevice));

    // --- Grid size
    dim3 dimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y);
    dim3 dimGrid (iDivUp(NX, BLOCK_SIZE_X), iDivUp(NY, BLOCK_SIZE_Y));

    // --- Jacobi iterations on the host
	Jacobi_Iterator_CPU(h_T, h_T_old, NX, NY, MAX_ITER);

	// --- Jacobi iterations on the device
    for (int k=0; k<MAX_ITER; k=k+2) {
        Jacobi_Iterator_GPU<<<dimGrid, dimBlock>>>(d_T,     d_T_old, NX, NY);   // --- Update d_T_old     starting from data stored in d_T
        Jacobi_Iterator_GPU<<<dimGrid, dimBlock>>>(d_T_old, d_T    , NX, NY);   // --- Update d_T         starting from data stored in d_T_old
    }

    // --- Copy result from device to host
    gpuErrchk(hipMemcpy(h_T_GPU_result, d_T, NX * NY * sizeof(float), hipMemcpyDeviceToHost));

	// --- Calculate percentage root mean square error between host and device results
	float sum = 0., sum_ref = 0.;
	for (int j=0; j<NY; j++)
		for (int i=0; i<NX; i++) {
			sum     = sum     + (h_T_GPU_result[j * NX + i] - h_T[j * NX + i]) * (h_T_GPU_result[j * NX + i] - h_T[j * NX + i]);
			sum_ref = sum_ref + h_T[j * NX + i]								   * h_T[j * NX + i];
		}
	printf("Percentage root mean square error = %f\n", 100.*sqrt(sum / sum_ref));
	
	// --- Release host memory 
    free(h_T);
	free(h_T_GPU_result);

    // --- Release device memory
    gpuErrchk(hipFree(d_T));
    gpuErrchk(hipFree(d_T_old));

    return 0;
}
