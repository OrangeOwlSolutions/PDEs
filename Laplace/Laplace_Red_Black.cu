#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include <hip/hip_runtime.h>

#include <thrust/reduce.h>
#include <thrust/device_vector.h>

#include "Utilities.cuh"
#include "InputOutput.h"
#include "TimingGPU.cuh"

// --- Problem size along one size. The computational domain is squared. NUM x NUM is the number of points in the
//     interior of the computational domain.
#define NUM			1024

#define BLOCKSIZEX	16
#define BLOCKSIZEY	16

// --- Double precision
#define DOUBLE

#ifdef DOUBLE
#define Real double
#define ZERO	0.0
#define ONE		1.0
#define TWO		2.0
#define FOUR	4.0

// --- SOR relaxation parameter
const Real omega = 1.85;
#else
#define Real	float
#define ZERO	0.0f
#define ONE		1.0f
#define TWO		2.0f
#define FOUR	4.0f

// --- SOR relaxation parameter
const Real omega = 1.85f;
#endif

// --- Split temperature into red and black arrays 
//#define MEMOPT

// --- Use texture memory
//#define TEXTURE

#ifdef TEXTURE
#ifdef DOUBLE
texture<int2, 1> t_aP;
texture<int2, 1> t_aW;
texture<int2, 1> t_aE;
texture<int2, 1> t_aS;
texture<int2, 1> t_aN;
texture<int2, 1> t_b;

static __inline__ __device__ double texFetch(texture<int2, 1> tex, int i)
{
	int2 v = tex1Dfetch(tex, i);
	return __hiloint2double(v.y, v.x);
}
#else
texture<float> t_aP;
texture<float> t_aW;
texture<float> t_aE;
texture<float> t_aS;
texture<float> t_aN;
texture<float> t_b;

static __inline__ __device__ float texFetch(texture<float> tex, int i)
{
	return tex1Dfetch(tex, i);
}
#endif
#endif

/*****************************/
/* SET EQUATION COEFFICIENTS */
/*****************************/
void setEquationCoefficients(const int Nrows, const int Ncols, const Real TN, Real * __restrict h_aP, Real * __restrict h_aW, Real * __restrict h_aE,
	Real * __restrict h_aS, Real * __restrict h_aN, Real * __restrict h_b)
{
	for (int col = 0; col < Ncols; ++col) {
		for (int row = 0; row < Nrows; ++row) {
			
			int ind = col * Nrows + row;

			h_b[ind]	= ZERO;

			// --- Left boundary condition: temperature is zero
			if (col == 0)			h_aW[ind] = ZERO;
			else					h_aW[ind] = ONE;

			// --- Right boundary condition: temperature is zero
			if (col == Ncols - 1)	h_aE[ind] = ZERO;
			else					h_aE[ind] = ONE;

			// --- Bottom boundary condition: temperature is zero
			if (row == 0)			h_aS[ind] = ZERO;
			else					h_aS[ind] = ONE;

			// --- Top boundary condition: temperature is TN
			if (row == Nrows - 1) {
									h_aN[ind] = ZERO;
									h_b[ind]  = FOUR * TN;
			}
			else					h_aN[ind] = ONE;

			h_aP[ind] = FOUR;
		}
	} 
} 

/********************************/
/* RED KERNEL - NO OPTIMIZATION */
/********************************/
template<class T>
__global__ void redKernelNoOptimization(const T * __restrict__ h_aP, const T * __restrict__ h_aW, const T * __restrict__ h_aE,
										const T * __restrict__ h_aS, const T * __restrict__ h_aN, const T * __restrict__ h_b,
										const T * __restrict__ h_tempBlack, T * __restrict__ h_tempRed, const T omega,
										T * __restrict__ norm_L2)
{
	// --- Addressing the interior of the (NUM + 2) x (NUM + 2) region
	const int tidx = blockIdx.x * blockDim.x + threadIdx.x + 1;			
	const int tidy = blockIdx.y * blockDim.y + threadIdx.y + 1;

	if ((tidx + tidy) % 2 != 0) return;									// --- If we are not on a "red" pixel, then exit.
		
	int ind_red = ( tidy      * (NUM + 2)) + tidx;						// --- Index for the "red" image
	int ind		= ((tidy - 1) * NUM)       + tidx - 1;					// --- Index for the coefficients

	T temp_old = h_tempRed[ind_red];

	T res = h_b[ind]
			+ (h_aW[ind] * h_tempBlack[tidx     + (tidy - 1) * (NUM + 2)]
			+ h_aE[ind]  * h_tempBlack[tidx     + (tidy + 1) * (NUM + 2)]
			+ h_aS[ind]  * h_tempBlack[tidx - 1 +  tidy      * (NUM + 2)]
			+ h_aN[ind]  * h_tempBlack[tidx + 1 +  tidy      * (NUM + 2)]);

	T temp_new = temp_old * (ONE - omega) + omega * (res / h_aP[ind]);

	h_tempRed[ind_red] = temp_new;
		
	res = temp_new - temp_old;
	norm_L2[ind_red] = res * res;

}

/**********************************/
/* BLACK KERNEL - NO OPTIMIZATION */
/**********************************/
template<class T>
__global__ void blackKernelNoOptimization(const T * __restrict__ h_aP, const T * __restrict__ h_aW, const T * __restrict__ h_aE,
	const T * __restrict__ h_aS, const T * __restrict__ h_aN, const T * __restrict__ h_b,
	const T * __restrict__ h_tempRed, T * __restrict__ h_tempBlack, const T omega,
	T * __restrict__ norm_L2)
{
	// --- Addressing the interior of the (NUM + 2) x (NUM + 2) region
	const int tidx = blockIdx.x * blockDim.x + threadIdx.x + 1;
	const int tidy = blockIdx.y * blockDim.y + threadIdx.y + 1;

	if ((tidx + tidy) % 2 == 0) return;									// --- If we are not on a "black" pixel, then exit.

	int ind_black = (tidy       * (NUM + 2)) + tidx;					// --- Index for the "black" image
	int ind = ((tidy - 1) * NUM) + tidx - 1;				// --- Index for the coefficients

	T temp_old = h_tempBlack[ind_black];

	T res = h_b[ind]
		+ (h_aW[ind] * h_tempRed[tidx + (tidy - 1) * (NUM + 2)]
		+ h_aE[ind] * h_tempRed[tidx + (tidy + 1) * (NUM + 2)]
		+ h_aS[ind] * h_tempRed[tidx - 1 + tidy      * (NUM + 2)]
		+ h_aN[ind] * h_tempRed[tidx + 1 + tidy      * (NUM + 2)]);

	//T res = h_b[ind]
	//	+ (h_aW[ind] * h_tempRed[tidx + (tidy - 1) * (NUM + 2)]);

	T temp_new = temp_old * (ONE - omega) + omega * (res / h_aP[ind]);

	h_tempBlack[ind_black] = temp_new;

	//h_tempBlack[ind_black] = temp_old;

	res = temp_new - temp_old;
	norm_L2[ind_black] = res * res;

}

//template<class T>
//__global__ void kernelNoOptimization(const T * __restrict__ h_aP, const T * __restrict__ h_aW, const T * __restrict__ h_aE,
//	const T * __restrict__ h_aS, const T * __restrict__ h_aN, const T * __restrict__ h_b,
//	const T * __restrict__ h_tempBlack, T * __restrict__ h_tempRed, const T omega, const T TN,
//	T * __restrict__ norm_L2)
//{
//	// --- Addressing the interior of the (NUM + 2) x (NUM + 2) region
//	const int tidx = blockIdx.x * blockDim.x + threadIdx.x;
//	const int tidy = blockIdx.y * blockDim.y + threadIdx.y;
//
//	//if ((tidx > NUM + 1) || (tidy > NUM + 1) || (tidx == 0) || (tidy == 0)) return;
//	if ((tidx > NUM + 1) || (tidy > NUM + 1)) return;
//
//	int ind_red = ( tidy      * (NUM + 2)) + tidx;						// --- Index for the "red" image
//	int ind		= ((tidy - 1) * NUM)       + tidx - 1;					// --- Index for the coefficients
//
//	T temp_old = h_tempRed[ind_red];
//
//	T res, temp_new;
//	
//	// --- Northern boundary condition
//	//if (tidy == 0) h_tempRed[ind_red] = TN;
//	//else if ((tidx == 0) || (tidx == NUM + 1) || (tidy == NUM + 1)) h_tempRed[ind_red] = ZERO;
//	//else {
//
//	    res = h_tempBlack[tidx + (tidy - 1) * (NUM + 2)] +
//			h_tempBlack[tidx + (tidy + 1) * (NUM + 2)] +
//			h_tempBlack[tidx - 1 + tidy      * (NUM + 2)] +
//			h_tempBlack[tidx + 1 + tidy      * (NUM + 2)];
//
//		temp_new = res / FOUR;
//
//		h_tempRed[ind_red] = temp_new;
//	//}
//
//	res = temp_new - temp_old;
//	norm_L2[ind_red] = res * res;
//
//}

/************************/
/* RED KERNEL - TEXTURE */
/************************/
#ifdef TEXTURE
template<class T>
__global__ void redKernelTexture(const T * __restrict__ h_tempBlack, T * __restrict__ h_tempRed, const T omega, T * __restrict__ norm_L2)
{
	// --- Addressing the interior of the (NUM + 2) x (NUM + 2) region
	const int tidx = blockIdx.x * blockDim.x + threadIdx.x + 1;
	const int tidy = blockIdx.y * blockDim.y + threadIdx.y + 1;

	if ((tidx + tidy) % 2 != 0) return;									// --- If we are not on a "red" pixel, then exit.
	
	int ind_red = ( tidy      * (NUM + 2)) + tidx;						// --- Index for the "red" image
	int ind		= ((tidy - 1) * NUM)       + tidx - 1;					// --- Index for the coefficients

	T temp_old = h_tempRed[ind_red];

	T res =  texFetch(t_b, ind)
			 + (texFetch(t_aW, ind) * h_tempBlack[tidx +     (tidy - 1) * (NUM + 2)]
			 +  texFetch(t_aE, ind) * h_tempBlack[tidx +     (tidy + 1) * (NUM + 2)]
			 +  texFetch(t_aS, ind) * h_tempBlack[tidx - 1 +  tidy      * (NUM + 2)]
			 +  texFetch(t_aN, ind) * h_tempBlack[tidx + 1 +  tidy      * (NUM + 2)]);
 
	T temp_new = temp_old * (ONE - omega) + omega * (res / texFetch(t_aP, ind));

	h_tempRed[ind_red] = temp_new;
	
	res = temp_new - temp_old;
	norm_L2[ind_red] = res * res;

}
#endif

/************************************/
/* RED KERNEL - MEMORY OPTIMIZATION */
/************************************/
template<class T>
__global__ void redKernelMemoryOptimization(const T * __restrict__ h_aP, const T * __restrict__ h_aW, const T * __restrict__ h_aE,
											const T * __restrict__ h_aS, const T * __restrict__ h_aN, const T * __restrict__ h_b,
											const T * __restrict__ h_tempBlack, T * __restrict__ h_tempRed, const T omega,
											T * __restrict__ norm_L2)
{
	// --- Addressing the interior of the (NUM + 2) x (NUM + 2) region
	const int tidx = blockIdx.x * blockDim.x + threadIdx.x + 1;
	const int tidy = blockIdx.y * blockDim.y + threadIdx.y + 1;

	int ind_red = tidy * ((NUM >> 1) + 2) + tidx;  				// --- Local (red) index; problemSize of the matrix is  ((NUM >> 1) + 2) x ((NUM >> 1) + 2)
	int ind		= 2 * tidx - (tidy & 1) - 1 + NUM * (tidy - 1);	// --- Global index

	T temp_old = h_tempRed[ind_red];

	T res = h_b[ind]
		    + (h_aW[ind] * h_tempBlack[tidx					+ (tidy - 1) * ((NUM >> 1) + 2)]
			+  h_aE[ind] * h_tempBlack[tidx					+ (tidy + 1) * ((NUM >> 1) + 2)]
			+  h_aS[ind] * h_tempBlack[tidx - (tidy & 1)		+	tidy	 * ((NUM >> 1) + 2)]
			+  h_aN[ind] * h_tempBlack[tidx + ((tidy + 1) & 1) +   tidy     * ((NUM >> 1) + 2)]);

	T temp_new = temp_old * (ONE - omega) + omega * (res / h_aP[ind]);

	h_tempRed[ind_red] = temp_new;
	res = temp_new - temp_old;

	norm_L2[ind_red] = res * res;
} 

/************************************************/
/* RED KERNEL - MEMORY OPTIMIZATION AND TEXTURE */
/************************************************/
#ifdef TEXTURE
template<class T>
__global__ void redKernelMemoryOptimizationTexture(const T * __restrict__ h_tempBlack, T * __restrict__ h_tempRed, const T omega, T * __restrict__ norm_L2)
{
	// --- Addressing the interior of the (NUM + 2) x (NUM + 2) region
	const int tidx = blockIdx.x * blockDim.x + threadIdx.x + 1;
	const int tidy = blockIdx.y * blockDim.y + threadIdx.y + 1;

	int ind_red = tidy * ((NUM >> 1) + 2) + tidx;  				// --- Local (red) index; problemSize of the matrix is  ((NUM >> 1) + 2) x ((NUM >> 1) + 2)
	int ind = 2 * tidx - (tidy & 1) - 1 + NUM * (tidy - 1);	// --- Global index

	T temp_old = h_tempRed[ind_red];

	T res = texFetch(t_b, ind)
			+ (texFetch(t_aW, ind) * h_tempBlack[tidx                    + (tidy - 1) * ((NUM >> 1) + 2)]
			+  texFetch(t_aE, ind) * h_tempBlack[tidx                    + (tidy + 1) * ((NUM >> 1) + 2)]
			+  texFetch(t_aS, ind) * h_tempBlack[tidx - (tidy & 1)       +  tidy      * ((NUM >> 1) + 2)]
			+  texFetch(t_aN, ind) * h_tempBlack[tidx + ((tidy + 1) & 1) +  tidy      * ((NUM >> 1) + 2)]);

	T temp_new = temp_old * (ONE - omega) + omega * (res / texFetch(t_aP, ind));

	h_tempRed[ind_red] = temp_new;
	res = temp_new - temp_old;

	norm_L2[ind_red] = res * res;
}
#endif

/**************************/
/* BLACK KERNEL - TEXTURE */
/**************************/
#ifdef TEXTURE
template<class T>
__global__ void blackKernelTexture(const T * __restrict__ h_tempRed, T * __restrict__ h_tempBlack, const T omega, T * __restrict__ norm_L2)
{
	// --- Addressing the interior of the (NUM + 2) x (NUM + 2) region
	const int tidx = blockIdx.x * blockDim.x + threadIdx.x + 1;
	const int tidy = blockIdx.y * blockDim.y + threadIdx.y + 1;

	if ((tidx + tidy) % 2 == 0) return;									// --- If we are not on a "black" pixel, then exit.
	
	int ind_black = (tidy       * (NUM + 2)) + tidx;					// --- Index for the "black" image
	int ind		  = ((tidy - 1) * NUM)       + tidx - 1;				// --- Index for the coefficients

	T temp_old = h_tempBlack[ind_black];

	T res =  texFetch(t_b, ind)
			 + (texFetch(t_aW, ind) * h_tempRed[tidx     + (tidy - 1) * (NUM + 2)]
			 + texFetch (t_aE, ind) * h_tempRed[tidx     + (tidy + 1) * (NUM + 2)]
			 + texFetch (t_aS, ind) * h_tempRed[tidx - 1 +  tidy      * (NUM + 2)]
			 + texFetch (t_aN, ind) * h_tempRed[tidx + 1 +  tidy      * (NUM + 2)]);

	T temp_new = temp_old * (ONE - omega) + omega * (res / texFetch(t_aP, ind));

	h_tempBlack[ind_black] = temp_new;
	res = temp_new - temp_old;

	norm_L2[ind_black] = res * res;

}
#endif

/**************************************/
/* BLACK KERNEL - MEMORY OPTIMIZATION */
/**************************************/
template<class T>
__global__ void blackKernelMemoryOptimization(const T * __restrict__ h_aP, const T * __restrict__ h_aW, const T * __restrict__ h_aE,
											  const T * __restrict__ h_aS, const T * __restrict__ h_aN, const T * __restrict__ h_b,
											  const T * __restrict__ h_tempRed, T * __restrict__ h_tempBlack, const T omega,
											  T * __restrict__ norm_L2)
{
	// --- Addressing the interior of the (NUM + 2) x (NUM + 2) region
	const int tidx = blockIdx.x * blockDim.x + threadIdx.x + 1;
	const int tidy = blockIdx.y * blockDim.y + threadIdx.y + 1;

	int ind_black = tidy * ((NUM >> 1) + 2) + tidx;  				// --- Local (red) index; problemSize of the matrix is  ((NUM >> 1) + 2) x ((NUM >> 1) + 2)
	int ind = 2 * tidx - ((tidy + 1) & 1) - 1 + NUM * (tidy - 1);	// --- Global index

	T temp_old = h_tempBlack[ind_black];

	T res = h_b[ind]
			+ (h_aW[ind] * h_tempRed[tidx						+ (tidy - 1) * ((NUM >> 1) + 2)]
			+  h_aE[ind] * h_tempRed[tidx						+ (tidy + 1) * ((NUM >> 1) + 2)]
			+  h_aS[ind] * h_tempRed[tidx - ((tidy + 1) & 1)	+ tidy		 * ((NUM >> 1) + 2)]
			+  h_aN[ind] * h_tempRed[tidx + (tidy & 1)			+ tidy		 * ((NUM >> 1) + 2)]);

	T temp_new = temp_old * (ONE - omega) + omega * (res / h_aP[ind]);

	h_tempBlack[ind_black] = temp_new;
	res = temp_new - temp_old;

	norm_L2[ind_black] = res * res;
} 

/**************************************************/
/* BLACK KERNEL - MEMORY OPTIMIZATION AND TEXTURE */
/**************************************************/
#ifdef TEXTURE
template<class T>
__global__ void blackKernelMemoryOptimizationTexture(const T * __restrict__ h_tempRed, T * __restrict__ h_tempBlack, const T omega, T * __restrict__ norm_L2)
{
	// --- Addressing the interior of the (NUM + 2) x (NUM + 2) region
	const int tidx = blockIdx.x * blockDim.x + threadIdx.x + 1;
	const int tidy = blockIdx.y * blockDim.y + threadIdx.y + 1;

	int ind_black = tidy * ((NUM >> 1) + 2) + tidx;  				// --- Local (red) index; problemSize of the matrix is  ((NUM >> 1) + 2) x ((NUM >> 1) + 2)
	int ind = 2 * tidx - ((tidy + 1) & 1) - 1 + NUM * (tidy - 1);	// --- Global index

	T temp_old = h_tempBlack[ind_black];

	T res = texFetch(t_b, ind)
			+ (texFetch(t_aW, ind) * h_tempRed[tidx                    + (tidy - 1) * ((NUM >> 1) + 2)]
			+  texFetch(t_aE, ind) * h_tempRed[tidx                    + (tidy + 1) * ((NUM >> 1) + 2)]
			+  texFetch(t_aS, ind) * h_tempRed[tidx - ((tidy + 1) & 1) +  tidy      * ((NUM >> 1) + 2)]
			+  texFetch(t_aN, ind) * h_tempRed[tidx +  (tidy & 1)      +  tidy      * ((NUM >> 1) + 2)]);

	T temp_new = temp_old * (ONE - omega) + omega * (res / texFetch(t_aP, ind));

	h_tempBlack[ind_black] = temp_new;
	res = temp_new - temp_old;

	norm_L2[ind_black] = res * res;
} 
#endif

/********/
/* MAIN */
/********/
int main(void) {

	TimingGPU timerGPU;
	
	/**********************/
	/* PROBLEM PARAMETERS */
	/**********************/
	Real TN				= 1.0;								// --- Temperature at northern boundary
	Real TS				= 0.;								// --- Temperature at southern boundary
	Real TW				= 0.;								// --- Temperature at western boundary	
	Real TE				= 0.;								// --- Temperature at eastern boundary

	//Real dx = L / NUM;																	// --- Discretization step along x-axis
	//Real dy = H / NUM;																	// --- Discretization step along y-axis

	// --- Number of discretization points along x and y including boundary points
#ifdef MEMOPT
	int Nrows = (NUM / 2) + 2;
#else
	int Nrows = NUM + 2;
#endif
	int Ncols = NUM + 2;
	
	// --- Problem size and computational size
	int problemSize			= NUM * NUM;
	int computationalSize	= Nrows * Ncols;

	/*************************/
	/* ITERATIONS PARAMETERS */
	/*************************/
	Real tol = 1.e-6;										// --- SOR iteration tolerance
	//int maxIter = 1e6;
	//int maxIter = 1e3;									// --- Maximum number of iterations
	int maxIter = 1;										// --- Maximum number of iterations

	int iter;

	/***************************/
	/* HOST MEMORY ALLOCATIONS */
	/***************************/
	// --- Equation coefficients
	Real *h_aP = (Real *)calloc(problemSize, sizeof(Real));						// --- Self coefficients
	Real *h_aW = (Real *)calloc(problemSize, sizeof(Real));						// --- West neighbor coefficients
	Real *h_aE = (Real *)calloc(problemSize, sizeof(Real));						// --- East neighbor coefficients
	Real *h_aS = (Real *)calloc(problemSize, sizeof(Real));						// --- South neighbor coefficients
	Real *h_aN = (Real *)calloc(problemSize, sizeof(Real));						// --- North neighbor coefficients

	// --- Right-hand side array
	Real *h_b = (Real *)calloc(problemSize, sizeof(Real));

	Real *h_tempRed		= (Real *)calloc(computationalSize, sizeof(Real));		// --- Red-cells temperature array
	Real *h_tempBlack	= (Real *)calloc(computationalSize, sizeof(Real));		// --- Black-cells temperature array

	// --- Set equation coefficients
	//setEquationCoefficients(NUM, NUM, thConductivity, dx, dy, width, TN, h_aP, h_aW, h_aE, h_aS, h_aN, h_b);
	setEquationCoefficients(NUM, NUM, TN, h_aP, h_aW, h_aE, h_aS, h_aN, h_b);

	/****************************/
	/* SET GRID AND BLOCK SIZES */
	/****************************/
	dim3 dimBlock(BLOCKSIZEX, BLOCKSIZEY);
#ifdef MEMOPT
	dim3 dimGrid(iDivUp(NUM / 2, BLOCKSIZEX), iDivUp(NUM, BLOCKSIZEY));
#else
	dim3 dimGrid(iDivUp(NUM, BLOCKSIZEX), iDivUp(NUM, BLOCKSIZEY));
	printf("dimGrid = %d %d\n", dimGrid.x, dimGrid.y);
#endif

	printf("Problem problemSize: %d x %d \n", NUM, NUM);

	timerGPU.StartCounter();

	/*****************************/
	/* DEVICE MEMORY ALLOCATIONS */
	/*****************************/
	Real *d_aP;			gpuErrchk(hipMalloc((void**)&d_aP, problemSize * sizeof(Real)));
	Real *d_aW;			gpuErrchk(hipMalloc((void**)&d_aW, problemSize * sizeof(Real)));
	Real *d_aE;			gpuErrchk(hipMalloc((void**)&d_aE, problemSize * sizeof(Real)));
	Real *d_aS;			gpuErrchk(hipMalloc((void**)&d_aS, problemSize * sizeof(Real)));
	Real *d_aN;			gpuErrchk(hipMalloc((void**)&d_aN, problemSize * sizeof(Real)));
	Real *d_b;			gpuErrchk(hipMalloc((void**)&d_b,  problemSize * sizeof(Real)));
	Real *d_tempRed;	gpuErrchk(hipMalloc((void**)&d_tempRed, computationalSize * sizeof(Real)));
#ifdef MEMOPT
	Real *d_tempBlack;	gpuErrchk(hipMalloc((void**)&d_tempBlack, computationalSize * sizeof(Real)));
#endif

	Real *d_L2DifferenceArray;	gpuErrchk(hipMalloc((void**)&d_L2DifferenceArray, computationalSize * sizeof(Real)));

	/*****************************/
	/* HOST-DEVICE MEMORY COPIES */
	/*****************************/
	gpuErrchk(hipMemcpy(d_aP, h_aP, problemSize * sizeof(Real), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_aW, h_aW, problemSize * sizeof(Real), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_aE, h_aE, problemSize * sizeof(Real), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_aS, h_aS, problemSize * sizeof(Real), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_aN, h_aN, problemSize * sizeof(Real), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_b, h_b, problemSize * sizeof(Real), hipMemcpyHostToDevice));
	gpuErrchk(hipMemset(d_tempRed, 0, computationalSize * sizeof(Real)));
#ifdef MEMOPT
	gpuErrchk(hipMemset(d_tempBlack, 0, computationalSize * sizeof(Real)));
#endif

	/********************/
	/* TEXTURE BINDINGS */
	/********************/
#ifdef TEXTURE
	gpuErrchk(hipBindTexture(NULL, t_aP, d_aP, problemSize * sizeof(Real)));
	gpuErrchk(hipBindTexture(NULL, t_aW, d_aW, problemSize * sizeof(Real)));
	gpuErrchk(hipBindTexture(NULL, t_aE, d_aE, problemSize * sizeof(Real)));
	gpuErrchk(hipBindTexture(NULL, t_aS, d_aS, problemSize * sizeof(Real)));
	gpuErrchk(hipBindTexture(NULL, t_aN, d_aN, problemSize * sizeof(Real)));
	gpuErrchk(hipBindTexture(NULL, t_b, d_b, problemSize * sizeof(Real)));
#endif

	/**************/
	/* ITERATIONS */
	/**************/
	for (iter = 0; iter < maxIter; ++iter) {

		// --- Update red cells
#if defined(TEXTURE) && defined(MEMOPT)
		redKernelMemoryOptimizationTexture << <dimGrid, dimBlock >> > (d_tempBlack, d_tempRed, omega, d_L2DifferenceArray);
#elif defined(TEXTURE) && !defined(MEMOPT)
		redKernelTexture << <dimGrid, dimBlock >> > (d_tempRed, d_tempRed, omega, d_L2DifferenceArray);
#elif !defined(TEXTURE) && defined(MEMOPT)
		redKernelMemoryOptimization << <dimGrid, dimBlock >> > (d_aP, d_aW, d_aE, d_aS, d_aN, d_b, d_tempBlack, d_tempRed, omega, d_L2DifferenceArray);
#else 
		redKernelNoOptimization << <dimGrid, dimBlock >> > (d_aP, d_aW, d_aE, d_aS, d_aN, d_b, d_tempRed, d_tempRed, omega, d_L2DifferenceArray);
#endif

		// --- Update black cells
#if defined(TEXTURE) && defined(MEMOPT)
		blackKernelMemoryOptimizationTexture << <dimGrid, dimBlock >> > (d_tempRed, d_tempBlack, omega, d_L2DifferenceArray);		
#elif defined(TEXTURE) && !defined(MEMOPT)
		blackKernelTexture << <dimGrid, dimBlock >> > (d_tempRed, d_tempRed, omega, d_L2DifferenceArray);
#elif !defined(TEXTURE) && defined(MEMOPT)
		blackKernelMemoryOptimization << <dimGrid, dimBlock >> > (d_aP, d_aW, d_aE, d_aS, d_aN, d_b, d_tempRed, d_tempBlack, omega, d_L2DifferenceArray);
#else 
		blackKernelNoOptimization << <dimGrid, dimBlock >> > (d_aP, d_aW, d_aE, d_aS, d_aN, d_b, d_tempRed, d_tempRed, omega, d_L2DifferenceArray);
#endif

		// --- Calculate residual
		Real norm_L2 = thrust::reduce(thrust::device_pointer_cast(d_L2DifferenceArray), thrust::device_pointer_cast(d_L2DifferenceArray) + computationalSize);
		norm_L2 = sqrt(norm_L2 / ((Real)problemSize));

		if (iter % 100 == 0) printf("%5d, %0.6f\n", iter, norm_L2);

		// --- If tolerance has been reached, end SOR iterations
		if (norm_L2 < tol) break;
	}

	// --- Transfer final red and black temperatures back to the host
	gpuErrchk(hipMemcpy(h_tempRed,   d_tempRed,   computationalSize * sizeof(Real), hipMemcpyDeviceToHost));
#ifdef MEMOPT
	//hipMemcpy(h_tempBlack, d_tempRed, computationalSize * sizeof(Real), hipMemcpyDeviceToHost);
	hipMemcpy(h_tempBlack, d_tempBlack, computationalSize * sizeof(Real), hipMemcpyDeviceToHost);
#endif

	double runtime = timerGPU.GetCounter();

	printf("GPU\n");
	printf("Iterations: %i\n", iter);
	printf("Total time: %f s\n", runtime / 1000.0);

	/****************************/
	/* SAVING THE FINAL RESULTS */
	/****************************/
	Real *h_T = (Real *)calloc(NUM * NUM, sizeof(Real));
	
	for (int row = 1; row < NUM + 1; ++row) {
		for (int col = 1; col < NUM + 1; ++col) {
			
			// --- Red cells
			if ((row + col) % 2 == 0) {
				int ind = col * Nrows + (row + (col % 2)) / 2;
				h_T[((col - 1) * NUM) + row - 1] = h_tempRed[ind];
			}
			// --- Black cells
			else {
				int ind = col * Nrows + (row + ((col + 1) % 2)) / 2;
#ifdef MEMOPT
				h_T[((col - 1) * NUM) + row - 1] = h_tempBlack[ind];
#else
				h_T[((col - 1) * NUM) + row - 1] = h_tempRed[ind];
#endif
			}
		}
	}

	printf("Saving...\n");
	saveCPUrealtxt(h_T, "D:\\Laplace\\Laplace_SOR_Red_Black\\Laplace_SOR_Red_Black\\Temp.txt", NUM * NUM);
	saveCPUrealtxt(h_tempRed, "D:\\Laplace\\Laplace_SOR_Red_Black\\Laplace_SOR_Red_Black\\Temp_red.txt", (NUM + 2) * (NUM + 2));

	// --- Free device memory
	gpuErrchk(hipFree(d_aP));
	gpuErrchk(hipFree(d_aW));
	gpuErrchk(hipFree(d_aE));
	gpuErrchk(hipFree(d_aS));
	gpuErrchk(hipFree(d_aN));
	gpuErrchk(hipFree(d_b));
	gpuErrchk(hipFree(d_tempRed));
#ifdef MEMOPT
	gpuErrchk(hipFree(d_tempBlack));
#endif

	gpuErrchk(hipFree(d_L2DifferenceArray));

#ifdef TEXTURE
	// --- Unbind textures
	gpuErrchk(hipUnbindTexture(t_aP));
	gpuErrchk(hipUnbindTexture(t_aW));
	gpuErrchk(hipUnbindTexture(t_aE));
	gpuErrchk(hipUnbindTexture(t_aS));
	gpuErrchk(hipUnbindTexture(t_aN));
	gpuErrchk(hipUnbindTexture(t_b));
#endif

	free(h_aP);
	free(h_aW);
	free(h_aE);
	free(h_aS);
	free(h_aN);
	free(h_b);
	free(h_tempRed);
	free(h_tempBlack);

	gpuErrchk(hipDeviceReset());

	return 0;
}
